#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cmath>
#include <algorithm>
#include <vector>
#include <sstream>
#include <fstream>
#include <chrono>
#include <cstdlib>
#include <cfloat>
#include <random>

using namespace std;

#define MAX_ITERATIONS 10
#define NUM_EXECUTIONS 1
#define NUM_ARGUMENTS 3


struct Point {
    double x;
    double y;
};

class KMedoidsData {
private:
    int numPoints; // Number of points in the dataset
    int numDimensions = 2; // Number of dimensions for each point
    vector<Point> points; // Vector of points

public:
    // Constructor that takes the file path and loads the points
    KMedoidsData(const string& filename) {
        loadPoints(filename);
    }

    // Method to load points from a file
    void loadPoints(const string& filename) {
        ifstream file(filename);
        if (!file.is_open()) {
            cerr << "Error opening file" << endl;
            return;
        }

        string line;
        numPoints = 0;
        numDimensions = 2; // We know the struct has 2 dimensions (x, y)

        while (getline(file, line)) {
            stringstream ss(line);
            string coord;
            double coords[2];
            int dimIdx = 0;

            // Parse each coordinate of the point
            while (getline(ss, coord, ',')) {
                if (dimIdx < 2) {
                    coords[dimIdx++] = stod(coord);
                }
                else {
                    cerr << "Error: More than two dimensions in the point" << endl;
                    return;
                }
            }

            if (dimIdx != 2) {
                cerr << "Error: Point does not have two dimensions" << endl;
                return;
            }

            points.push_back({ coords[0], coords[1] });
            numPoints++;
        }
        file.close();
    }

    // Method to get the points
    const vector<Point>& getPoints() const {
        return points;
    }

    // Method to get the number of points
    int getNumPoints() const {
        return numPoints;
    }

    // Method to get the number of dimensions
    int getNumDimensions() const {
        return numDimensions;
    }
};

void printCudaMemoryInfo(const char* stage) {
    size_t free_mem, total_mem;
    hipMemGetInfo(&free_mem, &total_mem);
    cout << stage << " - Free memory: " << free_mem / (1024 * 1024) << " MB, Total memory: " << total_mem / (1024 * 1024) << " MB" << endl;
}

// Kernel function to calculate the Euclidean distance between two points
__device__ double computeDistance(const Point p1, const Point p2) {
    return sqrt((p1.x - p2.x) * (p1.x - p2.x) + (p1.y - p2.y) * (p1.y - p2.y));

}

// Kernel function to calculate the nearest medoid for each point and update the cluster assignment
__global__ void calculateClusters(Point* d_p, int np, Point* d_m, int nc, int* d_c) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < np; i += blockDim.x * gridDim.x) {
        int min_index;
        double minDistance = DBL_MAX;

        for (int j = 0; j < nc; j++) {
            double distance = computeDistance(d_p[i], d_m[j]);
            if (distance < minDistance) {
                minDistance = distance;
                min_index = j;
            }
        }
        d_c[i] = min_index;
    }
}


__global__ void updateMedoids(Point* d_p, int np, Point* d_m, int nc, int* d_c, double* d_distances, int k) {

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < np; i += blockDim.x * gridDim.x) {
        int clusterIdx = d_c[i];
        Point p = d_p[i];
        int start = k;
        int end = k + gridDim.x*1000;
        double distance = 0.0;
        if (start > 0)
            distance = d_distances[i];
        // Calculate distances and find best medoids for each cluster
        for (int j = start; j < end && j < np; ++j) {
            if (d_c[j] == clusterIdx) {
                distance += computeDistance(p, d_p[j]);
            }
        }
        d_distances[i] = distance;
    }
}

void retrieveBestMedoids(Point* d_p, Point* d_m, int nc, int np, double* d_distances, vector<int> c) {
    // Host arrays to store intermediate results
    vector<double> SharedDistances(np);

    // Copy from device to host
    hipMemcpy(SharedDistances.data(), d_distances, np * sizeof(double), hipMemcpyDeviceToHost);

    // Final reduction on host
    vector<double> finalMinDistances(nc, DBL_MAX);
    vector<int> finalBestMedoids(nc, -1);

    for (int i = 0; i < np; ++i) {
        //cout << SharedDistances[i] << endl;
        if (SharedDistances[i] < finalMinDistances[c[i]]) {
            finalMinDistances[c[i]] = SharedDistances[i];
            finalBestMedoids[c[i]] = i;
            //cout << i << ": " << finalMinDistances[c[i]] << ", " << finalBestMedoids[c[i]] << endl;
        }
    }

    // Update medoids
    for (int i = 0; i < nc; ++i) {
        if (finalBestMedoids[i] != -1) {
            hipMemcpy(&d_m[i], &d_p[finalBestMedoids[i]], sizeof(Point), hipMemcpyDeviceToDevice);
        }
    }
}

vector<Point> initializeMedoids(vector<Point> p, int np, int nc) {

    vector<Point> medoids;

    // Initialize medoids at random positions
    int* medoidsIndices = new int[np];
    for (int i = 0; i < np; ++i)
        medoidsIndices[i] = i;

    random_shuffle(medoidsIndices, medoidsIndices + np); // shuffle indices randomly

    // Assign points as medoids
    for (int i = 0; i < nc; ++i)
        medoids.push_back(p[i]);

    return medoids;
}

long long kMedoids(vector<Point> p, int np, int nc, int nt, int nb) {

    printCudaMemoryInfo("Start execution");

    int size_p = np * sizeof(Point);
    Point* d_p;
    hipMalloc((void**)&d_p, size_p);
    hipMemcpy(d_p, p.data(), size_p, hipMemcpyHostToDevice);

    // Launch the kernel with enough threads to cover all points
    int threadsPerBlock = nt;
    int blocksPerGrid = nb;
    //cout << threadsPerBlock << ", " << blocksPerGrid << endl;

    vector<Point> medoids = initializeMedoids(p, np, nc);
    cout << "Old Medoids" << endl;
    for (int i = 0; i < medoids.size(); i++) {
        cout << medoids[i].x << "," << medoids[i].y << endl;
    }

    int size_m = nc * sizeof(Point);
    Point* d_m;
    hipMalloc((void**)&d_m, size_m);
    hipMemcpy(d_m, medoids.data(), size_m, hipMemcpyHostToDevice);

    int size_c = np * sizeof(int);
    int* d_c;
    vector<int> cluster(np);
    hipMalloc((void**)&d_c, size_c);
    hipMemset(d_c, -1, size_c);

    // Allocate memory for intermediate results
    double* d_distances;
    hipMalloc(&d_distances, np * sizeof(double));
    hipMemset(d_distances, -1, np * sizeof(double));

    printCudaMemoryInfo("After Creation");

    // Execute the K-Medoids algorithm on GPU
    auto start = chrono::steady_clock::now();

    int m = nb*1000 * ((np / (nb*1000)) + 1);

    for (int i = 0; i < MAX_ITERATIONS; i++) {
        calculateClusters << <blocksPerGrid, threadsPerBlock >> > (d_p, np, d_m, nc, d_c);
        hipDeviceSynchronize();

        hipMemcpy(cluster.data(), d_c, size_c, hipMemcpyDeviceToHost);
        /*for (int j = 0; j < np; j++) {
            if (cluster[j] == -1) {
                cout << j << " NO CLUSTER" << endl;
            }
        }*/

        for (int k = 0; k < m; k += nb*1000) {
            updateMedoids << <blocksPerGrid, threadsPerBlock >> > (d_p, np, d_m, nc, d_c, d_distances, k);
            hipDeviceSynchronize();
        }

        retrieveBestMedoids(d_p, d_m, nc, np, d_distances, cluster);
        hipMemcpy(medoids.data(), d_m, size_m, hipMemcpyDeviceToHost);


    }

    // Retrieve updated medoids from device to host
    hipMemcpy(medoids.data(), d_m, size_m, hipMemcpyDeviceToHost);

    auto end = chrono::steady_clock::now();
    auto duration = chrono::duration_cast<chrono::milliseconds>(end - start).count();

    cout << "Final Medoids" << endl;
    for (int i = 0; i < nc; i++) {
        cout << medoids[i].x << "," << medoids[i].y << endl;
    }

    hipFree(d_p);
    hipFree(d_m);
    hipFree(d_c);
    hipFree(d_distances);

    printCudaMemoryInfo("After deletion");

    return duration;
}

// Function to calculate the standard deviation of an array of values
// Parameters:
// - values: Array of values
// - mean: Mean value of the values array
// Returns:
// - Standard deviation of the values
long long standardDeviation(long long values[], long long mean) {
    long long std_deviation = 0;
    // Calculate the sum of squared differences from the mean
    for (int i = 0; i < NUM_EXECUTIONS; i++)
        std_deviation += pow(values[i] - mean, 2);

    std_deviation /= NUM_EXECUTIONS; // Divide by the number of values to get the mean of squared differences
    return sqrt(std_deviation); // Calculate the square root to get the standard deviation
}

// Function to find the minimum value in an array of values
// Parameters:
// - values: Array of values
// Returns:
// - Minimum value in the array
long long min_value(long long values[]) {
    long long minimum = values[0];
    // Iterate through the array to find the minimum value
    for (int i = 1; i < NUM_EXECUTIONS; i++)
        if (values[i] < minimum) // If the current value is less than the current minimum
            minimum = values[i]; // Update the minimum value

    return minimum; // Return the minimum value found
}

// Function to find the maximum value in an array of values
// Parameters:
// - values: Array of values
// Returns:
// - Maximum value in the array
long long max_value(long long values[]) {
    long long maximum = values[0];
    // Iterate through the array to find the maximum value
    for (int i = 1; i < NUM_EXECUTIONS; i++)
        if (values[i] > maximum) // If the current value is greater than the current maximum
            maximum = values[i]; // Update the maximum value

    return maximum; // Return the maximum value found
}

int main(int argc, char* argv[])
{
    /*if (argc != NUM_ARGUMENTS) {
        cout << "Wrong number of parameters" << endl;
        return -1;
    }*/

    int arr_np[] = { 1000 };
    int n_t[] = { 32 };
    int n_b[] = { 1, 2, 4, 8, 16, 32, 64, 128, 512, 1024 };
    // Parse command line arguments
    for (int i_np = 0; i_np < 10; i_np++) {
        for (int i_t = 0; i_t < 1; i_t++) {
                int np = arr_np[i_t]; // std::atoi(argv[1]); // Number of points
                int nc = 5;// std::atoi(argv[2]); // Number of clusters (K)

                // Specify the path to the file containing the points
                string filename = "../dataset/points" + to_string(np) + "_" + to_string(2) + "_" + to_string(nc) + ".txt";
                // Create an instance of KMedoidsData to load the points from the file
                KMedoidsData kmedoidsData(filename);
                long long total = 0.0;
                long long values[NUM_EXECUTIONS];

                const vector<Point>& p = kmedoidsData.getPoints();

                for (int i = 0; i < NUM_EXECUTIONS; i++) {
                    long long execution_time = kMedoids(p, np, nc, n_t[i_t], n_b[i_np]);
                    total += execution_time;
                    values[i] = execution_time;
                }
                // Calculate the arithmetic mean of execution times
                long long mean = total / NUM_EXECUTIONS;
                cout << np << "," << nc << "," << n_t[i_t] << "," << n_b[i_np] << "," << total << "," << mean << "," << standardDeviation(values, mean) << ',' << min_value(values) << ',' << max_value(values) << endl;

                // Prepare results in CSV format
                string output_filename = "results.csv";
                ofstream outfile(output_filename, ios::app); // Open file for appending
                if (!outfile.is_open()) {
                    cerr << "Error opening output file: " << output_filename << endl;
                    return -1;
                }

                // Write results to CSV file
                outfile << np << "," << nc << "," << n_t[i_t] << "," << n_b[i_np] << "," << total << "," << mean << ","
                    << standardDeviation(values, mean) << ',' << min_value(values) << ',' << max_value(values) << endl;

                // Close the file
                outfile.close();
        }
    }

    return 0;
}
